#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Constant memory for 3x3 kernel
__constant__ float cnn_kernel[9] = {0.0625, 0.125, 0.0625, 0.125, 0.25, 0.125, 0.0625, 0.125, 0.0625};

__global__ void cnnConvolution(float *input, float *output, int width, int height, int ksize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        float sum = 0;
        for (int ky = -ksize/2; ky <= ksize/2; ky++) {
            for (int kx = -ksize/2; kx <= ksize/2; kx++) {
                int imgX = x + kx, imgY = y + ky;
                if (imgX >= 0 && imgX < width && imgY >= 0 && imgY < height) {
                    sum += input[imgY * width + imgX] * cnn_kernel[(ky + ksize/2) * ksize + (kx + ksize/2)];
                }
            }
        }
        output[y * width + x] = sum;
    }
}

int main() {
    int width = 128, height = 128; // Spectrogram size
    int ksize = 3;
    float *h_input, *h_output, *d_input, *d_output;

    // Allocate host memory
    h_input = (float*)malloc(width * height * sizeof(float));
    h_output = (float*)malloc(width * height * sizeof(float));
    if (!h_input || !h_output) {
        printf("Host memory allocation failed\n");
        return 1;
    }

    // Initialize synthetic spectrogram
    srand(time(NULL));
    for (int i = 0; i < width * height; i++) h_input[i] = rand() / (float)RAND_MAX;

    // Allocate device memory
    hipError_t err;
    err = hipMalloc(&d_input, width * height * sizeof(float));
    err |= hipMalloc(&d_output, width * height * sizeof(float));
    if (err != hipSuccess) {
        printf("Device memory allocation failed: %s\n", hipGetErrorString(err));
        free(h_input); free(h_output);
        return 1;
    }

    // Copy data to device
    err = hipMemcpy(d_input, h_input, width * height * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Data transfer failed: %s\n", hipGetErrorString(err));
        hipFree(d_input); hipFree(d_output); free(h_input); free(h_output);
        return 1;
    }

    // Launch kernel
    dim3 threads(16, 16);
    dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);
    cnnConvolution<<<blocks, threads>>>(d_input, d_output, width, height, ksize);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("Kernel time: %f ms\n", ms);

    // Check for kernel errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    }

    // Copy results back
    err = hipMemcpy(h_output, d_output, width * height * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Result transfer failed: %s\n", hipGetErrorString(err));
    }

    // Save output
    FILE *fp = fopen("cnn_output.dat", "w");
    for (int i = 0; i < width * height; i++) {
        fprintf(fp, "%f\n", h_output[i]);
    }
    fclose(fp);

    // Free memory
    hipFree(d_input); hipFree(d_output);
    free(h_input); free(h_output);
    hipEventDestroy(start); hipEventDestroy(stop);
    return 0;
}
