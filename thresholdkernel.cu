#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void threshold(float *input, int *output, int n, float thresh) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx] = input[idx] > thresh ? 1 : 0; // 1 = fake, 0 = real
    }
}

int main() {
    int n = 1000; // Number of predictions
    float *h_input, *h_output, *d_input;
    int *d_output;

    // Allocate host memory
    h_input = (float*)malloc(n * sizeof(float));
    h_output = (int*)malloc(n * sizeof(int));
    if (!h_input || !h_output) {
        printf("Host memory allocation failed\n");
        return 1;
    }

    // Initialize synthetic probabilities
    srand(time(NULL));
    for (int i = 0; i < n; i++) h_input[i] = rand() / (float)RAND_MAX;

    // Allocate device memory
    hipError_t err;
    err = hipMalloc(&d_input, n * sizeof(float));
    err |= hipMalloc(&d_output, n * sizeof(int));
    if (err != hipSuccess) {
        printf("Device memory allocation failed: %s\n", hipGetErrorString(err));
        free(h_input); free(h_output);
        return 1;
    }

    // Copy data to device
    err = hipMemcpy(d_input, h_input, n * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Data transfer failed: %s\n", hipGetErrorString(err));
        hipFree(d_input); hipFree(d_output); free(h_input); free(h_output);
        return 1;
    }

    // Launch kernel
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);
    threshold<<<blocks, threads>>>(d_input, d_output, n, 0.5);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("Kernel time: %f ms\n", ms);

    // Check for kernel errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    }

    // Copy results back
    err = hipMemcpy(h_output, d_output, n * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Result transfer failed: %s\n", hipGetErrorString(err));
    }

    // Print sample results
    for (int i = 0; i < 10; i++) {
        printf("Prediction %d: %d\n", i, h_output[i]);
    }

    // Free memory
    hipFree(d_input); hipFree(d_output);
    free(h_input); free(h_output);
    hipEventDestroy(start); hipEventDestroy(stop);
    return 0;
}
