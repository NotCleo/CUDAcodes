
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define CHECK_CUFFT(err) if (err != HIPFFT_SUCCESS) { printf("cuFFT error: %d\n", err); exit(1); }

int main() {
    int n = 1048576; // 1M points
    hipfftComplex *h_data, *d_data;
    hipfftHandle plan;

    // Allocate host memory
    h_data = (hipfftComplex*)malloc(n * sizeof(hipfftComplex));
    if (!h_data) {
        printf("Host memory allocation failed\n");
        return 1;
    }

    // Initialize synthetic signal
    for (int i = 0; i < n; i++) {
        h_data[i].x = sin(2 * M_PI * i / 1024.0);
        h_data[i].y = 0;
    }

    // Allocate device memory
    hipError_t err;
    err = hipMalloc(&d_data, n * sizeof(hipfftComplex));
    if (err != hipSuccess) {
        printf("Device memory allocation failed: %s\n", hipGetErrorString(err));
        free(h_data);
        return 1;
    }

    // Copy data to device
    err = hipMemcpy(d_data, h_data, n * sizeof(hipfftComplex), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Data transfer failed: %s\n", hipGetErrorString(err));
        hipFree(d_data); free(h_data);
        return 1;
    }

    // Create cuFFT plan and execute
    CHECK_CUFFT(hipfftPlan1d(&plan, n, HIPFFT_C2C, 1));
    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);
    CHECK_CUFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("cuFFT time: %f ms\n", ms);

    // Copy results back
    err = hipMemcpy(h_data, d_data, n * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Result transfer failed: %s\n", hipGetErrorString(err));
    }

    // Print sample results
    for (int i = 0; i < 10; i++) {
        printf("Frequency %d: %f + %fi\n", i, h_data[i].x, h_data[i].y);
    }

    // Free memory
    CHECK_CUFFT(hipfftDestroy(plan));
    hipFree(d_data);
    free(h_data);
    hipEventDestroy(start); hipEventDestroy(stop);
    return 0;
}
