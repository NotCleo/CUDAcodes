#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void estimatePi(unsigned int *count, int points_per_thread, unsigned int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);
    unsigned int localCount = 0;
    for (int i = 0; i < points_per_thread; i++) {
        float x = hiprand_uniform(&state);
        float y = hiprand_uniform(&state);
        if (x * x + y * y <= 1.0f) localCount++;
    }
    count[idx] = localCount;
}

int main() {
    int n = 10000000, threads = 256, blocks = 100;
    int points_per_thread = n / (blocks * threads);
    unsigned int *h_count, *d_count;
    unsigned int total = 0;

    // Allocate memory
    h_count = (unsigned int*)malloc(blocks * threads * sizeof(unsigned int));
    hipError_t err = hipMalloc(&d_count, blocks * threads * sizeof(unsigned int));
    if (!h_count || err != hipSuccess) {
        printf("Memory allocation failed: %s\n", hipGetErrorString(err));
        free(h_count); hipFree(d_count);
        return 1;
    }

    // Launch kernel
    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);
    estimatePi<<<blocks, threads>>>(d_count, points_per_thread, (unsigned int)time(NULL));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("Kernel time: %f ms\n", ms);

    // Check for kernel errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    }

    // Copy results back
    err = hipMemcpy(h_count, d_count, blocks * threads * sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Result transfer failed: %s\n", hipGetErrorString(err));
    }

    // Sum counts
    for (int i = 0; i < blocks * threads; i++) total += h_count[i];
    printf("Pi ≈ %f\n", 4.0 * total / (float)n);

    // Free memory
    hipFree(d_count);
    free(h_count);
    hipEventDestroy(start); hipEventDestroy(stop);
    return 0;
}
