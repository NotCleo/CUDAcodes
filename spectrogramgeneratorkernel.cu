#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CHECK_CUFFT(err) if (err != HIPFFT_SUCCESS) { printf("cuFFT error: %d\n", err); exit(1); }

int main() {
    int n = 1024; // Window size for STFT
    int num_windows = 1000; // Number of segments
    hipfftComplex *h_data, *d_data;
    hipfftHandle plan;

    // Allocate host memory
    h_data = (hipfftComplex*)malloc(n * num_windows * sizeof(hipfftComplex));
    if (!h_data) {
        printf("Host memory allocation failed\n");
        return 1;
    }

    // Initialize synthetic audio
    for (int i = 0; i < n * num_windows; i++) {
        h_data[i].x = sin(2 * M_PI * i / 256.0); // Replace with audio samples
        h_data[i].y = 0;
    }

    // Allocate device memory
    hipError_t err;
    err = hipMalloc(&d_data, n * num_windows * sizeof(hipfftComplex));
    if (err != hipSuccess) {
        printf("Device memory allocation failed: %s\n", hipGetErrorString(err));
        free(h_data);
        return 1;
    }

    // Copy data to device
    err = hipMemcpy(d_data, h_data, n * num_windows * sizeof(hipfftComplex), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Data transfer failed: %s\n", hipGetErrorString(err));
        hipFree(d_data); free(h_data);
        return 1;
    }

    // Create batched cuFFT plan
    CHECK_CUFFT(hipfftPlan1d(&plan, n, HIPFFT_C2C, num_windows));
    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);
    CHECK_CUFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("cuFFT time: %f ms\n", ms);

    // Copy results back
    err = hipMemcpy(h_data, d_data, n * num_windows * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Result transfer failed: %s\n", hipGetErrorString(err));
    }

    // Save spectrogram (magnitude)
    FILE *fp = fopen("spectrogram.dat", "w");
    for (int i = 0; i < num_windows; i++) {
        for (int j = 0; j < n / 2; j++) {
            float mag = sqrt(h_data[i * n + j].x * h_data[i * n + j].x + h_data[i * n + j].y * h_data[i * n + j].y);
            fprintf(fp, "%f ", mag);
        }
        fprintf(fp, "\n");
    }
    fclose(fp);

    // Free memory
    CHECK_CUFFT(hipfftDestroy(plan));
    hipFree(d_data);
    free(h_data);
    hipEventDestroy(start); hipEventDestroy(stop);
    return 0;
}
