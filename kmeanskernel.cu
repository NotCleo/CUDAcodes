#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// CUDA kernel to assign points to nearest centroid
__global__ void assignClusters(float *points, float *centroids, int *assignments, int n, int k, int dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float minDist = 1e10;
        int minIdx = 0;
        for (int c = 0; c < k; c++) {
            float dist = 0;
            for (int d = 0; d < dim; d++) {
                float diff = points[idx * dim + d] - centroids[c * dim + d];
                dist += diff * diff;
            }
            if (dist < minDist) {
                minDist = dist;
                minIdx = c;
            }
        }
        assignments[idx] = minIdx;
    }
}

int main() {
    // Parameters
    int n = 100000, k = 10, dim = 2; // 100K 2D points, 10 clusters
    float *points, *centroids, *d_points, *d_centroids;
    int *assignments, *d_assignments;

    // Allocate host memory
    points = (float*)malloc(n * dim * sizeof(float));
    centroids = (float*)malloc(k * dim * sizeof(float));
    assignments = (int*)malloc(n * sizeof(int));
    if (!points || !centroids || !assignments) {
        printf("Host memory allocation failed\n");
        return 1;
    }

    // Initialize random data
    srand(time(NULL));
    for (int i = 0; i < n * dim; i++) points[i] = rand() / (float)RAND_MAX * 10;
    for (int i = 0; i < k * dim; i++) centroids[i] = rand() / (float)RAND_MAX * 10;

    // Allocate device memory
    hipError_t err;
    err = hipMalloc(&d_points, n * dim * sizeof(float));
    err |= hipMalloc(&d_centroids, k * dim * sizeof(float));
    err |= hipMalloc(&d_assignments, n * sizeof(int));
    if (err != hipSuccess) {
        printf("Device memory allocation failed: %s\n", hipGetErrorString(err));
        free(points); free(centroids); free(assignments);
        return 1;
    }

    // Copy data to device
    err = hipMemcpy(d_points, points, n * dim * sizeof(float), hipMemcpyHostToDevice);
    err |= hipMemcpy(d_centroids, centroids, k * dim * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Data transfer failed: %s\n", hipGetErrorString(err));
        hipFree(d_points); hipFree(d_centroids); hipFree(d_assignments);
        free(points); free(centroids); free(assignments);
        return 1;
    }

    // Launch kernel
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);
    assignClusters<<<blocks, threads>>>(d_points, d_centroids, d_assignments, n, k, dim);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("Kernel time: %f ms\n", ms);

    // Check for kernel errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    }

    // Copy results back
    err = hipMemcpy(assignments, d_assignments, n * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Result transfer failed: %s\n", hipGetErrorString(err));
    }

    // Print sample results
    for (int i = 0; i < 10; i++) {
        printf("Point %d assigned to cluster %d\n", i, assignments[i]);
    }

    // Free memory
    hipFree(d_points); hipFree(d_centroids); hipFree(d_assignments);
    free(points); free(centroids); free(assignments);
    hipEventDestroy(start); hipEventDestroy(stop);
    return 0;
}
