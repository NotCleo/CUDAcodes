#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Constant memory for 3x3 sharpening kernel
__constant__ float kernel[9] = {0, -1, 0, -1, 5, -1, 0, -1, 0};

__global__ void convolution(float *input, float *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        float sum = 0;
        for (int ky = -1; ky <= 1; ky++) {
            for (int kx = -1; kx <= 1; kx++) {
                int imgX = x + kx, imgY = y + ky;
                if (imgX >= 0 && imgX < width && imgY >= 0 && imgY < height) {
                    sum += input[imgY * width + imgX] * kernel[(ky + 1) * 3 + (kx + 1)];
                }
            }
        }
        output[y * width + x] = sum;
    }
}

int main() {
    int width = 1920, height = 1080;
    float *h_input, *h_output, *d_input, *d_output;

    // Allocate host memory
    h_input = (float*)malloc(width * height * sizeof(float));
    h_output = (float*)malloc(width * height * sizeof(float));
    if (!h_input || !h_output) {
        printf("Host memory allocation failed\n");
        return 1;
    }

    // Initialize synthetic image
    srand(time(NULL));
    for (int i = 0; i < width * height; i++) h_input[i] = rand() / (float)RAND_MAX;

    // Allocate device memory
    hipError_t err;
    err = hipMalloc(&d_input, width * height * sizeof(float));
    err |= hipMalloc(&d_output, width * height * sizeof(float));
    if (err != hipSuccess) {
        printf("Device memory allocation failed: %s\n", hipGetErrorString(err));
        free(h_input); free(h_output);
        return 1;
    }

    // Copy data to device
    err = hipMemcpy(d_input, h_input, width * height * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Data transfer failed: %s\n", hipGetErrorString(err));
        hipFree(d_input); hipFree(d_output); free(h_input); free(h_output);
        return 1;
    }

    // Launch kernel
    dim3 threads(16, 16);
    dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);
    convolution<<<blocks, threads>>>(d_input, d_output, width, height);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("Kernel time: %f ms\n", ms);

    // Check for kernel errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    }

    // Copy results back
    err = hipMemcpy(h_output, d_output, width * height * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Result transfer failed: %s\n", hipGetErrorString(err));
    }

    // Print sample results
    for (int i = 0; i < 10; i++) {
        printf("Pixel %d: %f\n", i, h_output[i]);
    }

    // Free memory
    hipFree(d_input); hipFree(d_output);
    free(h_input); free(h_output);
    hipEventDestroy(start); hipEventDestroy(stop);
    return 0;
}
